#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <cmath>
#include <hip/hip_runtime.h>

// CUDA kernel to calculate flashlight intensity
__global__ void calculateFlashlightIntensity(float* result, int width, int height, float x, float y, float radius, float intensityFactor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx < width && idy < height) {
        float dx = x - (idx / (float)width) * 2.0f + 1.0f;
        float dy = y - ((height - idy) / (float)height) * 2.0f + 1.0f;
        float distance = sqrtf(dx * dx + dy * dy);

        if (distance < radius) {
            result[index] = intensityFactor * (1.0f - distance / radius);
        } else {
            result[index] = 0.0f;
        }
    }
}

// CUDA calculator class to handle CUDA calculations
class CudaCalculator {
public:
    CudaCalculator(int screenWidth, int screenHeight)
        : screenWidth(screenWidth), screenHeight(screenHeight) {
        hipMalloc(&cudaIntensity, screenWidth * screenHeight * sizeof(float));
    }

    ~CudaCalculator() {
        hipFree(cudaIntensity);
    }

    // Calculate intensity using CUDA
    void calculateIntensity(float mouseX, float mouseY, float radius, float intensityFactor) {
        int gridSize = 16;
        dim3 block(gridSize, gridSize);
        dim3 grid((screenWidth + gridSize - 1) / gridSize, (screenHeight + gridSize - 1) / gridSize);

        calculateFlashlightIntensity<<<grid, block>>>(cudaIntensity, screenWidth, screenHeight, mouseX, mouseY, radius, intensityFactor);

        hipMemcpy(intensity, cudaIntensity, screenWidth * screenHeight * sizeof(float), hipMemcpyDeviceToHost);
    }

    // Get calculated intensity array
    float* getIntensity() {
        return intensity;
    }

private:
    const int screenWidth;
    const int screenHeight;

    float* cudaIntensity;
    float intensity[800 * 600];
};

// FlashlightApp class to manage the application
class FlashlightApp {
public:
    FlashlightApp()
        : screenWidth(800), screenHeight(600), radius(0.1f), intensityFactor(0.5f), isMouseInsideWindow(false) {
        mouseX = 0.0f;
        mouseY = 0.0f;
        cudaCalculator = new CudaCalculator(screenWidth, screenHeight);
    }

    ~FlashlightApp() {
        delete cudaCalculator;
    }

    // Initialize OpenGL settings
    void init() {
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);
        glMatrixMode(GL_MODELVIEW);
        glLoadIdentity();
    }

    // Display function to render OpenGL scene
    void display() {
        glClear(GL_COLOR_BUFFER_BIT);

        if (isMouseInsideWindow) {
            cudaCalculator->calculateIntensity(mouseX, mouseY, radius, intensityFactor);
            float* intensity = cudaCalculator->getIntensity();

            for (int y = 0; y < screenHeight; y++) {
                for (int x = 0; x < screenWidth; x++) {
                    int index = y * screenWidth + x;
                    glColor3f(intensity[index], 0.0f, 0.0f);
                    glBegin(GL_POINTS);
                    glVertex2f((x / (float)screenWidth) * 2.0f - 1.0f, ((screenHeight - y) / (float)screenHeight) * 2.0f - 1.0f);
                    glEnd();
                }
            }
        }

        glutSwapBuffers();
    }

    // Motion function to handle mouse movement
    void motion(int x, int y) {
        mouseX = (x / (float)screenWidth) * 2.0f - 1.0f;
        mouseY = ((screenHeight - y) / (float)screenHeight) * 2.0f - 1.0f;

        isMouseInsideWindow = (mouseX >= -1.0f && mouseX <= 1.0f && mouseY >= -1.0f && mouseY <= 1.0f);

        glutPostRedisplay();
    }

    // Run the application
    void run(int argc, char** argv) {
        appInstance = this; // Set the app instance
        glutInit(&argc, argv);
        glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
        glutInitWindowSize(screenWidth, screenHeight);
        glutCreateWindow("Flashlight Effect");

        glutDisplayFunc(displayWrapper);
        glutPassiveMotionFunc(motionWrapper);

        init();
        glutMainLoop();
    }

private:
    const int screenWidth;
    const int screenHeight;
    const float radius;
    const float intensityFactor;

    float mouseX;
    float mouseY;
    bool isMouseInsideWindow;

    CudaCalculator* cudaCalculator;
    static FlashlightApp* appInstance;

    static void displayWrapper() {
        appInstance->display();
    }

    static void motionWrapper(int x, int y) {
        appInstance->motion(x, y);
    }
};

FlashlightApp* FlashlightApp::appInstance = nullptr;

int main(int argc, char** argv) {
    FlashlightApp app;
    app.run(argc, argv);
    return 0;
}
