#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ 
void calculateFlashlightIntensity(float* cudaIntensity, int width, int height, float x, float y, float radius, float intensityFactor){

    const int idx = blockIdx.x*gridDim.x + threadIdx.x ;
    const int idy = blockIdx.y*gridDim.y + threadIdx.y ; 
    const int index = idx + idy*width;

    if (idx < width && idy < height) {
        float dx = x - (idx / (float)width) * 2.0f + 1.0f;
        float dy = y - ((height - idy) / (float)height) * 2.0f + 1.0f;
        float distance = sqrtf(dx * dx + dy * dy);

        if (distance < radius) {
            cudaIntensity[index] = intensityFactor * (1.0f - distance / radius);
        } else {
            cudaIntensity[index] = 0.0f;
        }
    }


}


__host__ 
Kernel::Kernel(int width, int height ) : Width(width), Height(height) {

        hipMalloc((void**)&cudaIntensity, Width*Height*sizeof(float)) ;
}


__host__
Kernel::~Kernel() {

    hipFree(cudaIntensity) ;

}

__host__ 
float * Kernel:: getIntensityAddress() {
    return hostIntensity;
}

__host__   
void Kernel::cudaLauncher(float x, float y, float radius, float intensityFactor) {
    
    dim3 threadSize(constant::threads, constant::threads) ;
    dim3 blockSize((Width + constant::threads - 1)/constant::threads, (Height + constant::threads - 1)/constant::threads) ;
    calculateFlashlightIntensity<<<blockSize,threadSize>>>(cudaIntensity,Width, Height,x,y,radius,intensityFactor) ;

}


