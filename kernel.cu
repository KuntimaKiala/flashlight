#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ 
void calculateFlashlightIntensity(float* cudaIntensity, int width, int height, float x, float y, float radius, float intensityFactor){

    const int idx = blockIdx.x*gridDim.x + threadIdx.x ;
    const int idy = blockIdx.y*gridDim.y + threadIdx.y ; 
    const int index = idx + idy*width;

    if (idx < width && idy < height) {
        float dx = x - (idx / (float)width) * 2.0f + 1.0f;
        float dy = y - ((height - idy) / (float)height) * 2.0f + 1.0f;
        float distance = sqrtf(dx * dx + dy * dy);


        if (distance < radius) {
            cudaIntensity[index] = intensityFactor * (1.0f - distance / radius);
        } else {
            cudaIntensity[index] = 0.0f;
        }
    }


}



__host__ 
Kernel::Kernel(int width, int height ) : Width(width), Height(height) {
    //float* intensity = new float[Width * Height];
    hipMalloc(&cudaIntensity, Width*Height*sizeof(float))  ;
}  


__host__
Kernel::~Kernel() {
    //delete [] intensity ;
    hipFree(cudaIntensity);
   
}

 float * Kernel::getIntensity() {
  
    return hostIntensity;
;
 }



__host__   
void Kernel::cudaLauncher(float x, float y, float radius, float intensityFactor) {
    
    dim3 block(constant::threads, constant::threads) ;
    dim3 grid( (Width + block.x - 1)/block.x, (Height + block.y - 1)/block.y) ;

   
    std::cout << Width << " " << Height << std::endl ;
    
    
    calculateFlashlightIntensity<<<grid,block>>>(cudaIntensity,Width, Height,x,y,radius,intensityFactor) ;
    

    hipMemcpy(hostIntensity, cudaIntensity, Height*Width* sizeof(float), hipMemcpyDeviceToHost);
    
    
    debugging("bro") ;
    //hipFree(cudaIntensity);
    

}


